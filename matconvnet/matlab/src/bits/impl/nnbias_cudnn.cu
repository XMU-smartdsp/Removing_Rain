// @file nnbias_cudnn.cu
// @brief biasolution block CuDNN-based implementation.
// @author Andrea Vedaldi

/*
Copyright (C) 2015 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#if !defined(ENABLE_GPU) | !defined(ENABLE_CUDNN)
#error "nnbias_cudnn.hpp cannot be compiled without GPU and CUDNN support."
#endif

#include "nnbias_cudnn.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include<iostream>

using namespace vl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
  error = context.setError(context.getCudaHelper().catchCudnnError(cudnnError, \
     STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
  goto done ; \
} }

/* ---------------------------------------------------------------- */
/*                                             nnbias_forward_cudnn */
/* ---------------------------------------------------------------- */

template<> vl::Error
vl::impl::nnbias_forward_cudnn<float>(vl::Context& context,
                                      vl::Tensor output, double outputMult,
                                      vl::Tensor data, double dataMult,
                                      vl::Tensor biases, double biasesMult)
{
  hipdnnTensorDescriptor_t outputDesc, biasesDesc, dataDesc ;
  bool outputDescInitialized = false ;
  bool biasesDescInitialized = false ;
  bool dataDescInitialized = false ;

  hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
  vl::Error error = vl::vlSuccess ;
  hipdnnHandle_t handle ;

  // Get CuDNN
  CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

  // Get output tensor descripotr
  assert(output) ;
  CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
  outputDescInitialized = true ;
  CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                   HIPDNN_TENSOR_NCHW,
                                   HIPDNN_DATA_FLOAT,
                                   output.getSize(), // sizes
                                   output.getDepth(),
                                   output.getWidth(),
                                   output.getHeight())) ;

  if (biases) {
    CHECK(hipdnnCreateTensorDescriptor(&biasesDesc)) ;
    biasesDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(biasesDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     1,
                                     biases.getNumElements(),
                                     1,
                                     1)) ;

    float alpha = biasesMult ;
    float beta = outputMult ;
    CHECK(hipdnnAddTensor(handle,
                         CUDNN_ADD_SAME_C,
                         &alpha,
                         biasesDesc, biases.getMemory(),
                         &beta,
                         outputDesc, output.getMemory())) ;
    outputMult = 1 ;
  }

  if (data) {
    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     data.getSize(),
                                     data.getDepth(),
                                     data.getWidth(),
                                     data.getHeight())) ;

    float alpha = dataMult ;
    float beta = outputMult ;
    CHECK(hipdnnAddTensor(handle,
                         CUDNN_ADD_FULL_TENSOR,
                         &alpha,
                         dataDesc, data.getMemory(),
                         &beta,
                         outputDesc, output.getMemory()));
  }

  /* cleanup */
done:
  if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
  if (biasesDescInitialized) { hipdnnDestroyTensorDescriptor(biasesDesc) ; }
  if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
  return context.passError(error, "nnbias_forward_cudnn: ") ;
}

/* ---------------------------------------------------------------- */
/*                                            nnbias_backward_cudnn */
/* ---------------------------------------------------------------- */

template<> vl::Error
vl::impl::nnbias_backward_cudnn<float>(vl::Context& context,
                                       vl::Tensor derData, double derDataMult,
                                       vl::Tensor derBiases, double derBiasesMult,
                                       vl::Tensor derOutput, double derOutputMult)
{

  /* no derDataDesc needed as same as dataDesc */
  hipdnnTensorDescriptor_t derDataDesc, derBiasesDesc, derOutputDesc ;
  bool derDataDescInitialized = false ;
  bool derBiasesDescInitialized = false ;
  bool derOutputDescInitialized = false ;


  hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
  vl::Error error = vl::vlSuccess ;
  hipdnnHandle_t handle ;

  // Get CuDNN
  CHECK(context.getCudaHelper().getCudnnHandle(&handle)) ;

  // Must have derOutput for all derivatives
  assert(derOutput) ;
  CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
  derOutputDescInitialized = true ;
  CHECK(hipdnnSetTensor4dDescriptor(derOutputDesc,
                                   HIPDNN_TENSOR_NCHW,
                                   HIPDNN_DATA_FLOAT,
                                   derOutput.getSize(), // sizes
                                   derOutput.getDepth(),
                                   derOutput.getWidth(),
                                   derOutput.getHeight())) ;

  // for derivatives w.r.t. bias
  if (derBiases) {
    CHECK(hipdnnCreateTensorDescriptor(&derBiasesDesc)) ;
    derBiasesDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derBiasesDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     1,
                                     derBiases.getNumElements(),
                                     1,
                                     1)) ;

    float alpha = derOutputMult ;
    float beta = derBiasesMult ;
    CHECK(hipdnnConvolutionBackwardBias
          (handle,
           &alpha,
           derOutputDesc, derOutput.getMemory(),
           &beta,
           derBiasesDesc, derBiases.getMemory())) ;
  }

  if (derData) {
    CHECK(hipdnnCreateTensorDescriptor(&derDataDesc)) ;
    derDataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derDataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     derData.getSize(),
                                     derData.getDepth(),
                                     derData.getWidth(),
                                     derData.getHeight())) ;
    // not implemented
    assert(false) ;
  }

done:
  if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
  if (derBiasesDescInitialized) { hipdnnDestroyTensorDescriptor(derBiasesDesc) ; }
  if (derDataDescInitialized) { hipdnnDestroyTensorDescriptor(derDataDesc) ; }
  return context.passError(error, "nnbias_backward_cudnn: ") ;
}

